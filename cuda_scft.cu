#include "hip/hip_runtime.h"
#include"struct.h"

#include "init_cuda.h"
#include "cuda_scft.h"
#include <errno.h>

#include <typeinfo>
#include"cuda_aid.cuh"





extern void average_value(std::vector<double*> data,GPU_INFO *gpu_info,CUFFT_INFO *cufft_info){

	int gpu_index;
	
	int threads=gpu_info->thread;
	
	size_t smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

	std::vector<double*> sum;

	sum.resize(gpu_info->GPU_N);

		
	
	for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){	

		checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));

		checkCudaErrors(hipMallocManaged((void**)&(sum[gpu_index]), sizeof(double)* cufft_info->batch));
		
		reduce3<double><<< cufft_info->batch, threads, smemSize,gpu_info->stream[gpu_index] >>>(data[gpu_index], sum[gpu_index], cufft_info->NxNyNz);
	
		
	}
	
	
	dim3 block(cufft_info->Nx,cufft_info->Ny,cufft_info->Nz);
	
	for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){
		
		checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));	
		
		minus_average<<<block,cufft_info->batch,0,gpu_info->stream[gpu_index]>>>(data[gpu_index],sum[gpu_index]);
		
		
	}
	for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){	

		checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));	

		checkCudaErrors(hipFree(sum[gpu_index]));
		
		checkCudaErrors(hipDeviceSynchronize());
	}	
	
}

extern void getConc(GPU_INFO *gpu_info,CUFFT_INFO *cufft_info){

	int gpu_index;

	dim3 grid(cufft_info->Nx,cufft_info->Ny,cufft_info->Nz);

	int threads=512;

	size_t smemSize = threads * sizeof(double)*2;//(threads <= 32) ? 2 * threads * sizeof(double) : 

	average_value(cufft_info->wa_cu,gpu_info,cufft_info);	

	for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){
		
		checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));	
	
		qInt_init<<<grid,cufft_info->batch,0,gpu_info->stream[gpu_index]>>>(cufft_info->qInt_cu[gpu_index]);

		checkCudaErrors(hipDeviceSynchronize());

	}

	sovDifFft(gpu_info,cufft_info,cufft_info->qa_cu,cufft_info->wa_cu,cufft_info->NsA,1);
	
	
	sovDifFft(gpu_info,cufft_info,cufft_info->qcb_cu,cufft_info->wb_cu,cufft_info->dNsB,-1);


	for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){
		
		checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));	

		qa_to_qInt<<<grid,cufft_info->batch,0,gpu_info->stream[gpu_index]>>>(cufft_info->qInt_cu[gpu_index],cufft_info->qa_cu[gpu_index],cufft_info->NsA);

	}
	

	
	sovDifFft(gpu_info,cufft_info,cufft_info->qb_cu,cufft_info->wb_cu,cufft_info->dNsB,1);

	
	for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){
		
		checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));	

		qa_to_qInt2<<<grid,cufft_info->batch,0,gpu_info->stream[gpu_index]>>>(cufft_info->qInt_cu[gpu_index],cufft_info->qcb_cu[gpu_index],cufft_info->dNsB);
		
		checkCudaErrors(hipDeviceSynchronize());

	}
	
	sovDifFft(gpu_info,cufft_info,cufft_info->qca_cu,cufft_info->wa_cu,cufft_info->NsA,-1);
	//for(int i=0;i<20;i++) printf("%g\n",cufft_info->qca_cu[0][i*cufft_info->NsA]);
	for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){
		
		checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));	

		cal_ql<<<cufft_info->batch,threads,smemSize,gpu_info->stream[gpu_index]>>>(cufft_info->ql[gpu_index],cufft_info->qb_cu[gpu_index],cufft_info->dNsB,cufft_info->NxNyNz);

		checkCudaErrors(hipDeviceSynchronize());
		checkCudaErrors(hipGetLastError());

		for(int i=0;i<cufft_info->batch;i++){
			cufft_info->ql[gpu_index][i]/=cufft_info->NxNyNz;
			cufft_info->ffl[gpu_index][i]=cufft_info->ds0/cufft_info->ql[gpu_index][i];
		}
		
		//w_to_phi<<<grid,cufft_info->batch,0,gpu_info->stream[gpu_index]>>>(cufft_info->pha_cu[gpu_index], cufft_info->phb_cu[gpu_index],cufft_info->qa_cu[gpu_index],cufft_info->qca_cu[gpu_index],cufft_info->qb_cu[gpu_index],cufft_info->qcb_cu[gpu_index],cufft_info->NsA,cufft_info->dNsB,cufft_info->ffl[gpu_index]);
		dim3 gridgo(cufft_info->NxNyNz/gpu_info->thread,cufft_info->batch);
		w_to_phi_go<<<gridgo,gpu_info->thread,0,gpu_info->stream[gpu_index]>>>(cufft_info->pha_cu[gpu_index], cufft_info->phb_cu[gpu_index],cufft_info->qa_cu[gpu_index],cufft_info->qca_cu[gpu_index],cufft_info->qb_cu[gpu_index],cufft_info->qcb_cu[gpu_index],cufft_info->NsA,cufft_info->dNsB,cufft_info->ffl[gpu_index]);

		checkCudaErrors(hipDeviceSynchronize());
		
	//printf("cal=%g\n",cufft_info->ql[gpu_index][0]);
	}

	checkCudaErrors(hipGetLastError());
}

extern double Free(GPU_INFO *gpu_info,CUFFT_INFO *cufft_info){

	dim3 grid(cufft_info->Nx,cufft_info->Ny,cufft_info->Nz);

	double *freeEnergy,*freeOld;
	
	double *freeW,*freeAB,*freeS,*freeDiff,*freeWsurf;

	double *inCompMax,*fpsum,*psum;
	
	int iter=0;

	int i;

	int gpu_index;	

	freeEnergy=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	freeOld=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	freeW=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	freeAB=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	freeS=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	freeDiff=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	freeWsurf=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	inCompMax=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	fpsum=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	psum=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);

	do{
		iter=iter+1;
	
		average_value(cufft_info->wa_cu,gpu_info,cufft_info); 

		average_value(cufft_info->wb_cu,gpu_info,cufft_info);

		getConc(gpu_info,cufft_info);

	

		for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){

			checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));	
		
			phi_w<<<grid,cufft_info->batch,0,gpu_info->stream[gpu_index]>>>(cufft_info->wa_cu[gpu_index],cufft_info->wb_cu[gpu_index],cufft_info->pha_cu[gpu_index],cufft_info->phb_cu[gpu_index], cufft_info->hAB);
	
			checkCudaErrors(hipDeviceSynchronize());

		}

		if(iter%cufft_info->AverIt==0){

			for(i=0;i<cufft_info->batch*gpu_info->GPU_N;i++){	
				freeW[i]=0.0;
				freeAB[i]=0.0;
				freeS[i]=0.0;
				freeWsurf[i]=0.0;
				inCompMax[i]=0.0;
			}// end i 

		
			for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++)
				for(i=0;i<cufft_info->batch;i++){
				
					for(long ijk=0;ijk<cufft_info->NxNyNz;ijk++){
						psum[i+gpu_index*cufft_info->batch]=1-cufft_info->pha_cu[gpu_index][ijk+i*cufft_info->NxNyNz]-cufft_info->phb_cu[gpu_index][ijk+i*cufft_info->NxNyNz];
						fpsum[i+gpu_index*cufft_info->batch]=fabs(psum[i+gpu_index*cufft_info->batch]);

						if(fpsum[i+gpu_index*cufft_info->batch]>inCompMax[i+gpu_index*cufft_info->batch]) inCompMax[i+gpu_index*cufft_info->batch]=fpsum[i+gpu_index*cufft_info->batch];
						freeAB[i+gpu_index*cufft_info->batch]=freeAB[i+gpu_index*cufft_info->batch]+cufft_info->hAB*cufft_info->pha_cu[gpu_index][ijk+i*cufft_info->NxNyNz]*cufft_info->phb_cu[gpu_index][ijk+i*cufft_info->NxNyNz];
						freeW[i+gpu_index*cufft_info->batch]=freeW[i+gpu_index*cufft_info->batch]-(cufft_info->wa_cu[gpu_index][ijk+i*cufft_info->NxNyNz]*cufft_info->pha_cu[gpu_index][ijk+i*cufft_info->NxNyNz]+cufft_info->wb_cu[gpu_index][ijk+i*cufft_info->NxNyNz]*cufft_info->phb_cu[gpu_index][ijk+i*cufft_info->NxNyNz]);
				
					}
				
					freeAB[i+gpu_index*cufft_info->batch]/=cufft_info->NxNyNz;
					//printf("freeW=%0.10f\n",freeW);
					freeW[i+gpu_index*cufft_info->batch]/=cufft_info->NxNyNz;
					freeWsurf[i+gpu_index*cufft_info->batch]/=cufft_info->NxNyNz;
				
					freeS[i+gpu_index*cufft_info->batch]=-log(cufft_info->ql[gpu_index][i]);
					//printf("%d %.10f %.10f %.10f %.10f\n",i,qCab[0],qCab[1],freeS[i],-log(qCab[1]));
					freeOld[i+gpu_index*cufft_info->batch]=freeEnergy[i+gpu_index*cufft_info->batch];
					freeEnergy[i+gpu_index*cufft_info->batch]=freeAB[i+gpu_index*cufft_info->batch]+freeW[i+gpu_index*cufft_info->batch]+freeS[i+gpu_index*cufft_info->batch];
					printf("GPU %d batch %d: %5d : %.8e, %.8e, %.8e,%.8e, %.8e\n", gpu_index,i,iter, freeEnergy[i+gpu_index*cufft_info->batch],freeAB[i+gpu_index*cufft_info->batch],freeW[i+gpu_index*cufft_info->batch], freeS[i+gpu_index*cufft_info->batch],inCompMax[i+gpu_index*cufft_info->batch]);
				
				}// end for i


			
			for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++)
				for(i=0;i<cufft_info->batch;i++){
		
					FILE *dp;
		
					char filename[20];
					if(iter%(cufft_info->AverIt*10)==0){
						sprintf(filename,"pha_%d.dat",gpu_index*cufft_info->batch+i+1);

						dp=fopen(filename,"w");
						fprintf(dp,"Nx=%d, Ny=%d, Nz=%d",cufft_info->Nx,cufft_info->Ny,cufft_info->Nz);
						fprintf(dp,"dx=%d, dy=%d, dz=%d",cufft_info->dx,cufft_info->dy,cufft_info->dz);
						for(int ijk=0;ijk<cufft_info->NxNyNz;ijk++)
						fprintf(dp,"%g %g %g %g\n",cufft_info->pha_cu[gpu_index][ijk+i*cufft_info->NxNyNz],cufft_info->phb_cu[gpu_index][ijk+i*cufft_info->NxNyNz],cufft_info->wa_cu[gpu_index][ijk+i*cufft_info->NxNyNz],cufft_info->wb_cu[gpu_index][ijk+i*cufft_info->NxNyNz]);

						fclose(dp);
					}
	

				}
		}// end for if Aver It
	
		

		

	}while(iter<cufft_info->MaxIT);//! end loop do


	free(freeEnergy);
	free(freeOld);
	free(freeW);
	free(freeAB);
	free(freeS);
	free(freeDiff);
	free(freeWsurf);
	free(inCompMax);
	free(psum);
	free(fpsum);
	
	return 0;

}

extern double Free_um(GPU_INFO *gpu_info,CUFFT_INFO *cufft_info){

	dim3 grid(cufft_info->Nx,cufft_info->Ny,cufft_info->Nz);

	double *freeEnergy,*freeOld;
	
	double *freeW,*freeAB,*freeS,*freeDiff,*freeWsurf;

	double *inCompMax,*fpsum,*psum;
	
	int iter=0;

	int i;

	int gpu_index;	

	freeEnergy=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	freeOld=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	freeW=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	freeAB=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	freeS=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	freeDiff=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	freeWsurf=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	inCompMax=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	fpsum=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);
	psum=(double*)malloc(sizeof(double)*cufft_info->batch*gpu_info->GPU_N);

	do{
		iter=iter+1;
	
		average_value(cufft_info->wa_cu,gpu_info,cufft_info); 

		average_value(cufft_info->wb_cu,gpu_info,cufft_info);

		getConc(gpu_info,cufft_info);

	

		for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){

			checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));	
		
			phi_w<<<grid,cufft_info->batch,0,gpu_info->stream[gpu_index]>>>(cufft_info->wa_cu[gpu_index],cufft_info->wb_cu[gpu_index],cufft_info->pha_cu[gpu_index],cufft_info->phb_cu[gpu_index], cufft_info->hAB);
	
			checkCudaErrors(hipDeviceSynchronize());

		}

		if(iter%cufft_info->AverIt==0){

			for(i=0;i<cufft_info->batch*gpu_info->GPU_N;i++){	
				freeW[i]=0.0;
				freeAB[i]=0.0;
				freeS[i]=0.0;
				freeWsurf[i]=0.0;
				inCompMax[i]=0.0;
			}

		
			for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++)
				for(i=0;i<cufft_info->batch;i++){
				
					for(long ijk=0;ijk<cufft_info->NxNyNz;ijk++){
						psum[i+gpu_index*cufft_info->batch]=1-cufft_info->pha_cu[gpu_index][ijk+i*cufft_info->NxNyNz]-cufft_info->phb_cu[gpu_index][ijk+i*cufft_info->NxNyNz];
						fpsum[i+gpu_index*cufft_info->batch]=fabs(psum[i+gpu_index*cufft_info->batch]);

						if(fpsum[i+gpu_index*cufft_info->batch]>inCompMax[i+gpu_index*cufft_info->batch]) inCompMax[i+gpu_index*cufft_info->batch]=fpsum[i+gpu_index*cufft_info->batch];
						freeAB[i+gpu_index*cufft_info->batch]=freeAB[i+gpu_index*cufft_info->batch]+cufft_info->hAB*cufft_info->pha_cu[gpu_index][ijk+i*cufft_info->NxNyNz]*cufft_info->phb_cu[gpu_index][ijk+i*cufft_info->NxNyNz];
						freeW[i+gpu_index*cufft_info->batch]=freeW[i+gpu_index*cufft_info->batch]-(cufft_info->wa_cu[gpu_index][ijk+i*cufft_info->NxNyNz]*cufft_info->pha_cu[gpu_index][ijk+i*cufft_info->NxNyNz]+cufft_info->wb_cu[gpu_index][ijk+i*cufft_info->NxNyNz]*cufft_info->phb_cu[gpu_index][ijk+i*cufft_info->NxNyNz]);
				
					}
				
					freeAB[i+gpu_index*cufft_info->batch]/=cufft_info->NxNyNz;
					//printf("freeW=%0.10f\n",freeW);
					freeW[i+gpu_index*cufft_info->batch]/=cufft_info->NxNyNz;
					freeWsurf[i+gpu_index*cufft_info->batch]/=cufft_info->NxNyNz;
				
					freeS[i+gpu_index*cufft_info->batch]=-log(cufft_info->ql[gpu_index][i]);
					//printf("%d %.10f %.10f %.10f %.10f\n",i,qCab[0],qCab[1],freeS[i],-log(qCab[1]));
					freeOld[i+gpu_index*cufft_info->batch]=freeEnergy[i+gpu_index*cufft_info->batch];
					freeEnergy[i+gpu_index*cufft_info->batch]=freeAB[i+gpu_index*cufft_info->batch]+freeW[i+gpu_index*cufft_info->batch]+freeS[i+gpu_index*cufft_info->batch];
					printf("GPU %d batch %d: %5d : %.8e, %.8e, %.8e,%.8e, %.8e\n", gpu_index,i,iter, freeEnergy[i+gpu_index*cufft_info->batch],freeAB[i+gpu_index*cufft_info->batch],freeW[i+gpu_index*cufft_info->batch], freeS[i+gpu_index*cufft_info->batch],inCompMax[i+gpu_index*cufft_info->batch]);
				
				}// end for i
			}// end for gpu_index
	
		

		

	}while(iter<cufft_info->MaxIT);//! end loop do


	free(freeEnergy);
	free(freeOld);
	free(freeW);
	free(freeAB);
	free(freeS);
	free(freeDiff);
	free(freeWsurf);
	free(inCompMax);
	free(psum);
	free(fpsum);
	
	return 0;

}




extern void fft_test(GPU_INFO *gpu_info,CUFFT_INFO *cufft_info){

	//int gpu_index;

	//long NxNyNz,ijk;

	//NxNyNz=cufft_info->NxNyNz;

	hipEvent_t start,stop;

	float msec;

	hipError_t error;

	//int dNsB=cufft_info->dNsB;
	
	//int threads=gpu_info->thread;
	
	//size_t smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);
/*
	for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){	

			checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));
			if(gpu_index==0)	
			for(long ijk=0;ijk<10;ijk++) printf("%g \n",cufft_info->wb_cu[gpu_index][ijk]);
		
	}
*/
	
	error=hipEventCreate(&start);
	error=hipEventCreate(&stop);
	error=hipEventCreate(&start);
	error=hipEventCreate(&stop);
	error=hipEventRecord(start,0);	
		//getConc(gpu_info,cufft_info);
		Free(gpu_info,cufft_info);
	error=hipEventRecord(stop,0);	
	hipEventSynchronize(stop);	
			
	error=hipEventElapsedTime(&msec,start,stop);

	if(error!=hipSuccess) printf("fft_test did not successfully detect run time\n");
			
	printf("time=%0.10f\n",msec);
	
	
	
	
	

}


extern void sovDifFft(GPU_INFO *gpu_info,CUFFT_INFO *cufft_info,std::vector<double*> g,std::vector<double*> w,int ns,int sign){
	
	int ns1=ns+1;
	int Nx=cufft_info->Nx;
	int Ny=cufft_info->Ny;
	int Nz=cufft_info->Nz;
	int gpu_index;	
	int iz;
	
	dim3 grid(Nx,Ny,Nz),block(cufft_info->batch,1,1),grid1(cufft_info->Nxh1,Ny,Nz);
	
	for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){	
		
		checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));
		
		initilize_wdz<<<grid,block,0,gpu_info->stream[gpu_index]>>>(w[gpu_index],cufft_info->wdz_cu[gpu_index],cufft_info->ds2);

		

	}
	
	

	if(sign==1){
		for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){	

			checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));

			initilize_q<<<grid,block,1,gpu_info->stream[gpu_index]>>>(g[gpu_index],cufft_info->qInt_cu[gpu_index],ns1);//,gpu_info->stream[gpu_index]
			
		}

		for(iz=1;iz<=ns;iz++){
			for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){	
		
				checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));
				dim3 gridgo(cufft_info->NxNyNz/gpu_info->thread,cufft_info->batch);

				initilize_in_go<<<gridgo,gpu_info->thread,0,gpu_info->stream[gpu_index]>>>(cufft_info->device_in[gpu_index],g[gpu_index],cufft_info->wdz_cu[gpu_index],ns1,iz);
				//initilize_in<<<grid,block,0,gpu_info->stream[gpu_index]>>>(cufft_info->device_in[gpu_index],g[gpu_index],cufft_info->wdz_cu[gpu_index],ns1,iz);
			
				checkCudaErrors(hipfftExecD2Z(cufft_info->plan_forward[gpu_index],cufft_info->device_in[gpu_index],cufft_info->device_out[gpu_index]));
				
				
				//sufaceField<<<grid1,block,0,gpu_info->stream[gpu_index]>>>(cufft_info->device_out[gpu_index],cufft_info->kxyzdz_cu[gpu_index],cufft_info->Nx);
				dim3 gridgo_sur(cufft_info->Nxh1NyNz/gpu_info->thread_sur,cufft_info->batch);

				sufaceField_go<<<gridgo_sur,gpu_info->thread_sur,0,gpu_info->stream[gpu_index]>>>(cufft_info->device_out[gpu_index],cufft_info->kxyzdz_cu[gpu_index],cufft_info->Nxh1,cufft_info->Nx,cufft_info->Ny,cufft_info->Nz);
		
				checkCudaErrors(hipfftExecZ2D(cufft_info->plan_backward[gpu_index],cufft_info->device_out[gpu_index],cufft_info->device_in[gpu_index]));
				
				in_to_g_go<<<gridgo,gpu_info->thread,0,gpu_info->stream[gpu_index]>>>(g[gpu_index],cufft_info->wdz_cu[gpu_index],cufft_info->device_in[gpu_index],ns1, iz);
				//in_to_g<<<grid,block,0,gpu_info->stream[gpu_index]>>>(g[gpu_index],cufft_info->wdz_cu[gpu_index],cufft_info->device_in[gpu_index],ns1, iz);
				checkCudaErrors(hipDeviceSynchronize());
				checkCudaErrors(hipGetLastError()); 	
				
			}
	

		}
		for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){	
	
			checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));
			
			checkCudaErrors(hipStreamSynchronize(gpu_info->stream[gpu_index]));

			checkCudaErrors(hipDeviceSynchronize());
			
		}
		
	}
	else if(sign==-1){
		
	
		for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){	
	
			checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));

			initilize_q_inverse<<<grid,block,0,gpu_info->stream[gpu_index]>>>(g[gpu_index],cufft_info->qInt_cu[gpu_index],ns1);//,gpu_info->stream[gpu_index]
			
			
			
		}
		
		
	
		for(iz=ns-1;iz>=0;iz--){
		
			for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){	
	
				checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));
				dim3 gridgo(cufft_info->NxNyNz/gpu_info->thread,cufft_info->batch);
				//initilize_in<<<grid,block,0,gpu_info->stream[gpu_index]>>>(cufft_info->device_in[gpu_index],g[gpu_index],cufft_info->wdz_cu[gpu_index],ns1,iz+2);
				initilize_in_go<<<gridgo,gpu_info->thread,0,gpu_info->stream[gpu_index]>>>(cufft_info->device_in[gpu_index],g[gpu_index],cufft_info->wdz_cu[gpu_index],ns1,iz+2);
				
				checkCudaErrors(hipfftExecD2Z(cufft_info->plan_forward[gpu_index],cufft_info->device_in[gpu_index],cufft_info->device_out[gpu_index]));

				//sufaceField<<<grid1,block,0,gpu_info->stream[gpu_index]>>>(cufft_info->device_out[gpu_index],cufft_info->kxyzdz_cu[gpu_index],cufft_info->Nx);
				dim3 gridgo_sur(cufft_info->Nxh1NyNz/gpu_info->thread_sur,cufft_info->batch);

				sufaceField_go<<<gridgo_sur,gpu_info->thread_sur,0,gpu_info->stream[gpu_index]>>>(cufft_info->device_out[gpu_index],cufft_info->kxyzdz_cu[gpu_index],cufft_info->Nxh1,cufft_info->Nx,cufft_info->Ny,cufft_info->Nz);
		
				checkCudaErrors(hipfftExecZ2D(cufft_info->plan_backward[gpu_index],cufft_info->device_out[gpu_index],cufft_info->device_in[gpu_index]));

				//in_to_g<<<grid,block,0,gpu_info->stream[gpu_index]>>>(g[gpu_index],cufft_info->wdz_cu[gpu_index],cufft_info->device_in[gpu_index],ns1, iz);
				in_to_g_go<<<gridgo,gpu_info->thread,0,gpu_info->stream[gpu_index]>>>(g[gpu_index],cufft_info->wdz_cu[gpu_index],cufft_info->device_in[gpu_index],ns1, iz);
				checkCudaErrors(hipDeviceSynchronize());
				checkCudaErrors(hipGetLastError()); 	
			}

		}
		for(gpu_index=0;gpu_index<gpu_info->GPU_N;gpu_index++){	
	
			checkCudaErrors(hipSetDevice(gpu_info->whichGPUs[gpu_index]));
			
			checkCudaErrors(hipStreamSynchronize(gpu_info->stream[gpu_index]));
			
			checkCudaErrors(hipDeviceSynchronize());
		}
		
		
	}
	
	
	
	
	

}












